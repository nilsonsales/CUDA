#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>

#define N 512

__global__ void add(int *a, int *b, int *c){
    c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];  // Use threadIdx.x for multiple threads

}


// Atribute random values to elements of a[n]
void random_ints(int *a, int n){
   for(int i = 0; i < n; ++i)
       a[i] = rand() %5000;
}

// Print all elements of a[n]
void printv(int *a, int n){
    printf("[ ");
    for(int i = 0; i < N; ++i){
        printf("%d ", a[i]);
    }
    printf("]\n");
}


int main(){
    int *a, *b, *c;  // Host copies of a, b, c
    int *dev_a, *dev_b, *dev_c;  // Device copies of a, b, c
    int size = N * sizeof(int);

    // Allocate space for device copies a, b, c
    hipMalloc((void **) &dev_a, size);
    hipMalloc((void **) &dev_b, size);
    hipMalloc((void **) &dev_c, size);

    // Allocate space for host copies of a, b, c and setup input values
    a = (int *) malloc(size);
    b = (int *) malloc(size);
    c = (int *) malloc(size);

    random_ints(a, N);
    random_ints(b, N);

    printv(a, N);
    printv(b, N);


    // Copy inputs to device
    hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);

    // Launch add kernel on GPU
    add<<<N,1>>>(dev_a, dev_b, dev_c);  // N blocks, 1 thread

    // Copy result back to host
    hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost);

    printv(c, N);

    // Clean up
    free(a); free(b); free(c);
    hipFree(dev_a); hipFree(dev_b); hipFree(dev_c);

    return 0;
}